
#include <hip/hip_runtime.h>
__global__ void sineWaveKernel(float4 *pos, unsigned int width, unsigned int height, float animtime)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0 - 1.0;
	v = v*2.0 - 1.0;

	float frequency = 4.0;
	float w = sinf(frequency*u + animtime)*cosf(frequency*v + animtime)*0.5;
	pos[y*width + x] = make_float4(u, w, v, 1.0);
}

void LaunchCudaKernal(float4 *pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(meshWidth / block.x, meshHeight / block.y, 1.0);
	sineWaveKernel <<<grid,block>>> (pos, meshWidth, meshHeight, time);
}
